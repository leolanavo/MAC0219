
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime_api.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}


void matSum(float* S, float* A, float* B, int N) {
for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      int tid = i*N+j;
      S[tid] = A[tid] + B[tid];
    }
  }    
}


// Fills a vector with random float entries.
void randomInit(float* data, int N) {
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      int tid = i*N+j;
      data[tid] = (float)drand48();
    }
  }
}


int main(int argc, char* argv[])
{

  if (argc != 2) {
    fprintf(stderr, "Syntax: %s <matrix size>  \n", argv[0]);
    return EXIT_FAILURE;
  }

  int N = atoi(argv[1]);

  // set seed for drand48()
  srand48(42);

  // allocate host memory for matrices A and B
  //printf("Allocate host memory for matrices A and B...\n");
  float* A = (float*) malloc(N * N * sizeof(float));
  float* B = (float*) malloc(N * N * sizeof(float));
  float* S = (float*) malloc(N * N * sizeof(float));

  // initialize host matrices
  //printf("Initialize host matrices...\n");
  randomInit(A, N);
  randomInit(B, N);

  // execute the kernel
  printf("Execute the kernel...\n");
  struct timeval begin, end;
  gettimeofday(&begin, NULL);

  matSum(S, A, B, N);

  gettimeofday(&end, NULL);
  float cpuTime = 1000000*(float)(end.tv_sec - begin.tv_sec);
  cpuTime +=  (float)(end.tv_usec - begin.tv_usec);
  // print times
  printf("\nExecution Time (microseconds): %9.2f\n\n", cpuTime);


  // clean up memory
  free(A);
  free(B);
  free(S);

  return 0;
}

