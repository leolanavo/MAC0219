
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <sstream>
#include <fstream>
#include <string>
#include <climits>
#include <cmath>

const int DIMENSION = 3;
const int BLOCK_SIZE = 1024;
const int BLOCK_ITEMS = 2048;
const int EXPONENT = 11;
const int DEVICE_ID = 0;
const int ELEMENTS = DIMENSION * DIMENSION;

void read_file(std::ifstream& file, const int number_of_matrices,
        const int number_of_matrices_std, std::vector<int>& v) {
    int first, second, third;
    std::string dump;

    for (int k = 0; k < number_of_matrices; k++) {
        file >> dump;

        for (int i = 0; i < ELEMENTS; i += DIMENSION) {
            file >> first >> second >> third;
            v[(number_of_matrices_std * i) + k] = first;
            v[(number_of_matrices_std * (i + 1)) + k] = second;
            v[(number_of_matrices_std * (i + 2)) + k] = third;
        }
    }
}

std::vector<int> set_host_v(char* filename) {
    int number_of_matrices;
    std::ifstream file(filename);

    if (file.is_open()) file >> number_of_matrices;

    int number_of_matrices_std = ceil(number_of_matrices / (double) BLOCK_ITEMS) * BLOCK_ITEMS;
    std::vector<int> v (ELEMENTS * number_of_matrices_std, INT_MAX);
    read_file(file, number_of_matrices, number_of_matrices_std, v);

    file.close();
    return v;
}

void set_device_v(int number_of_matrices, std::vector<int>& host_v, void*& device_v) {
    int block_size = ELEMENTS * number_of_matrices * sizeof(int);
    hipMalloc(&device_v, block_size);
    hipMemcpy(device_v, host_v.data(), block_size, hipMemcpyHostToDevice);
}

__global__ void reduce_block(void* m) {
    int* m_int = (int*) m;
    int index = (BLOCK_ITEMS * blockIdx.x) + threadIdx.x;
    __shared__ int m_shared[BLOCK_ITEMS];

    m_shared[threadIdx.x] = m_int[index];
    m_shared[BLOCK_SIZE + threadIdx.x] = m_int[BLOCK_SIZE + index];

    for (int block_size = BLOCK_ITEMS; block_size > 1; block_size >>= 1) {
        int index_1 = threadIdx.x;
        int index_2 = index_1 + (block_size / 2);

        if (threadIdx.x >= (block_size >> 1))
            return;

        m_shared[index_1] = min(m_shared[index_1], m_shared[index_2]);
        __syncthreads();
    }

    if (threadIdx.x == 0)
        m_int[index] = m_shared[0];
}

__global__ void compress_block(void* v, int number_of_matrices, int offset) {
    int* v_int = (int*) v;

    int index_compressed =
        (blockIdx.x * number_of_matrices) + threadIdx.x +
        (offset * BLOCK_SIZE);

    int index_expanded =
        (blockIdx.x * number_of_matrices) + threadIdx.x * BLOCK_ITEMS +
        (offset * BLOCK_ITEMS * BLOCK_SIZE);

    v_int[index_compressed] = v_int[index_expanded];
    __syncthreads();
}

__global__ void compress_line(void* v, int line,
        int number_of_blocks, int next_number_of_blocks) {

    int index_expanded_1 = line * number_of_blocks * BLOCK_ITEMS + threadIdx.x;
    int index_compressed_1 = line * next_number_of_blocks * BLOCK_ITEMS + threadIdx.x;

    int index_expanded_2 = line * number_of_blocks * BLOCK_ITEMS +
        BLOCK_SIZE + threadIdx.x;
    int index_compressed_2 = line * next_number_of_blocks * BLOCK_ITEMS +
        BLOCK_SIZE + threadIdx.x;

    int* v_int = (int*) v;
    v_int[index_compressed_1] = v_int[index_expanded_1];
    v_int[index_compressed_2] = v_int[index_expanded_2];
}

__global__ void final_compress(void* v, int number_of_matrices) {
    int* v_int = (int*) v;
    for (int i = 0; i < ELEMENTS; i++) {
        v_int[i] = v_int[i * BLOCK_ITEMS];
    }
}

void print_result_matrix(void* device_v) {
    int result[ELEMENTS];

    hipMemcpy((void*)result, device_v, ELEMENTS * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < ELEMENTS; i++) {
        std::cout << result[i];
        if ((i + 1) % DIMENSION) std::cout << " ";
        else std::cout << std::endl;
    }
}

int main(int argc, char* argv[]) {
    void* device_v;
    hipSetDevice(DEVICE_ID);

    std::vector<int> host_v = set_host_v(argv[1]);
    int number_of_matrices = host_v.size() / ELEMENTS;

    set_device_v(number_of_matrices, host_v, device_v);

    for (int k = number_of_matrices; k > 1; k >>= EXPONENT) {
        k = ceil(k / (double) BLOCK_ITEMS) * BLOCK_ITEMS;

        // Reduction
        int number_of_blocks = k >> EXPONENT;
        reduce_block<<< number_of_blocks * ELEMENTS, BLOCK_SIZE >>>(device_v);

        int next_number_of_blocks = ceil (number_of_blocks / (double) BLOCK_ITEMS);
        /* std::cout << next_number_of_blocks << " " << number_of_blocks << std::endl; */

        // Compression
        int threads_per_block = (number_of_blocks < BLOCK_SIZE)?
            number_of_blocks : BLOCK_SIZE;
        for (int i = 0; i < number_of_blocks >> 10; i++)
            compress_block<<< ELEMENTS, threads_per_block >>> (device_v, k, i);

        for (int i = 1; i < ELEMENTS && number_of_blocks > 1; i++)
            compress_line<<< 1, BLOCK_SIZE >>>
                (device_v, i, number_of_blocks, next_number_of_blocks);
    }

    final_compress<<<1,1>>>(device_v, number_of_matrices);
    print_result_matrix(device_v);

    hipDeviceReset();
    return(0);
}
