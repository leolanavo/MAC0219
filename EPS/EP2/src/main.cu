#include <vector>
#include <iostream>
#include <sstream>
#include <fstream>
#include <string>
#include <climits>

const int DIMENSION = 3;
const int BLOCK_SIE = 1024;
const int DEVICE_ID = 0;

#define Matrix std::vector<std::vector<int>>

std::ostream& operator<<(std::ostream& os, Matrix m) {
    for (auto line = m.begin(); line != m.end(); line++) {
        for (auto item = (*line).begin(); item != (*line).end(); item++) {
            os << *(item) << " ";
        }
        os << std::endl;
    }
    return os;
}

void add_infinite(Matrix& m) {
    for (int i = 0; i < DIMENSION*DIMENSION; i++)
        m[i].push_back(INT_MAX);
}

void read_file(std::ifstream& file, const int number_of_matrices, Matrix& m) {
    int first, second, third;
    std::string line;

    for (int k = 0; k < number_of_matrices; k++) {
        // This is used to throw away the line of asterisks
        file >> line;

        for (int i = 0; i < DIMENSION; i++) {
            file >> first >> second >> third;

            m[(i * DIMENSION)].push_back(first);
            m[(i * DIMENSION) + 1].push_back(second);
            m[(i * DIMENSION) + 2].push_back(third);
        }
    }
}

int set_host_matrix(std::ifstream& file, Matrix& m) {
    int number_of_matrices;

    if (input_file.is_open())
        input_file >> number_of_matrices;

    read_file(input_file, number_of_matrices, matrices_host);
    if (number_of_matrices % 2 != 0) add_infinite(matrices_host);

    return number_of_matrices;
}

void set_device_matrix(int number_of_matrices, Matrix& device_matrix, Matrix& host_matrix) {
    hipSetDevice(DEVICE_ID);

    for (int i = 0; i < DIMENSION * DIMENSION; i++) {
        hipMalloc((void**) matrices_device[i], number_of_matrices);
        hipMemcpy(device_matrix[i], host_matrix[i], hipMemcpyHostToDevice);
    }
}

int main(int argc, char* argv[]) {
    Matrix host_matrix (DIMENSION * DIMENSION);
    Matrix device_matrix (DIMENSION * DIMENSION);

    int number_of_matrices = set_host_matrix(std::ifstream(argv[1]), host_matrix);
    set_host_matrix(number_of_matrices, device_matrix, host_matrix);

    return(0);
}
